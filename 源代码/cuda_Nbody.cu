
#include <hip/hip_runtime.h>
#include <ctime>
#include <cstdio>

const float softeningFactor = 1e-8;

// 考虑物体j吸引物体i，增加物体i的对应分加速度
__device__ void addAcceleration(float4 &pi, float4 &pj, float3 &ai)
{
    // 获得位矢
    float3 distVector = {0.0f, 0.0f, 0.0f};
    distVector.x = pj.x - pi.x;
    distVector.y = pj.y - pi.y;
    distVector.z = pj.z - pi.z;

    // 计算标量部分
    float distSquared = distVector.x * distVector.x + distVector.y * distVector.y + distVector.z * distVector.z;
    distSquared += softeningFactor;
    float invDist = rsqrtf(distSquared);
    float invDistCubic = invDist * invDist * invDist;
    float coff = pj.w * invDistCubic;

    // 累加加速度分矢量
    ai.x += distVector.x * coff;
    ai.y += distVector.y * coff;
    ai.z += distVector.z * coff;
}

// 计算某个物体的总加速度
__device__ float3 calcGrossAcc(float4 &bodyPos, float4 *positions, int numBatches)
{
    extern __shared__ float4 sharedPos[]; // 某个批次的物体的位置，存放在共享内存中

    float3 acc = {0.0f, 0.0f, 0.0f};

    for (unsigned int batch = 0; batch < numBatches; ++batch) // 利用block分批次并行处理所有物体，每次读入线程数个物体，正好每个线程搬一个物体数据
    {
        sharedPos[threadIdx.x] = positions[batch * blockDim.x + threadIdx.x]; // 并行读入当前批次的物体位置到共享内存中
        __syncthreads();                                                      // 并行同步

        // tile calculation.
#pragma unroll 128
        for (unsigned int index = 0; index < blockDim.x; ++index) // 对于当前批次的所有物体，考虑其对于目标物体的加速度
        {
            addAcceleration(bodyPos, sharedPos[index], acc);
        }
        __syncthreads();
    }

    return acc;
}

// 已知上一时间的所有物体位置，根据万有引力公式计算下一时间的某个物体位置
__global__ void calcGrossPos(
    float4 *newPos,
    float4 *oldPos,
    float3 *velocity,
    float deltaTime,
    float damping,
    unsigned int numBatches,
    unsigned int numBodies) // 由于线程自动补足，numBodies没用。而且计算时，空物体无质量，不会贡献加速度。
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // 线程负责的物体

    float4 pos = oldPos[index];

    float3 accel = calcGrossAcc(pos, oldPos, numBatches); // 计算目标物体的下一个位置，block内的线程会协作转移数据，从而采用更高效的共享内存计算

    // 更新物体速度
    float3 vel = velocity[index];
    vel.x += accel.x * deltaTime;
    vel.y += accel.y * deltaTime;
    vel.z += accel.z * deltaTime;
    vel.x *= damping;
    vel.y *= damping;
    vel.z *= damping;

    // 更新物体位置
    pos.x += vel.x * deltaTime;
    pos.y += vel.y * deltaTime;
    pos.z += vel.z * deltaTime;

    // 放到新的外部数组中
    newPos[index] = pos;
    velocity[index] = vel;

    // 由于速度是与计算独立的，而新位置放到了新数组内，因此不用考虑并行同步问题
}

const float damping = 1;
const int blockSize = 256;
const int T = 1000;        // 计算轮数
const float deltaTime = 1; // 一秒一个时间点，方便计算
const float G = 1e5;

int main()
{
    // 主机端读入质点数据
    int n;
    float4 *hPos;
    float3 *hVel;
    freopen("data.in", "r", stdin);
    scanf("%d", &n);

    // 计算并行参数
    int numBlocks = (n + blockSize - 1) / blockSize; // 一个线程负责一个物体的加速度计算，那么物体数除以block大小就是block个数
    int sharedMemSize = blockSize * sizeof(float4);

    //继续读入和分配空间
    int n_new = numBlocks * blockSize;  //扩充数组，避免溢出
    size_t bytes4 = n_new * sizeof(float4);
    size_t bytes3 = n_new * sizeof(float3);
    hPos = (float4 *)malloc(bytes4);
    hVel = (float3 *)malloc(bytes3);
    for (int i = 0; i < n; ++i)
    {
        scanf("%f%f%f", &hPos[i].x, &hPos[i].y, &hPos[i].z);
        scanf("%f%f%f", &hVel[i].x, &hVel[i].y, &hVel[i].z);
        scanf("%f", &hPos[i].w);
        hPos[i].w *= G;
    }
    fclose(stdin);

    // 计时
    clock_t start = clock();

    // 设备端申请内存并复制数据
    float4 *dPos, *nPos;
    float3 *dVel;
    hipMalloc(&dPos, bytes4);
    hipMalloc(&dVel, bytes3);
    hipMemcpy(dPos, hPos, bytes4, hipMemcpyHostToDevice);
    hipMemcpy(dVel, hVel, bytes3, hipMemcpyHostToDevice);
    hipMalloc(&nPos, bytes4);

    // 创建历史记录数组
    float4 *history[T];

    // 开始每轮按顺序模拟
    for (int i = 0; i < T; ++i)
    {
        calcGrossPos<<<numBlocks, blockSize, sharedMemSize>>>(nPos, dPos, dVel, deltaTime, damping, numBlocks, n);
        hipDeviceSynchronize(); // 注意，在所有物体都被计算完后，才开始更新和输出位置
        hipMemcpy(dPos, nPos, bytes4, hipMemcpyDeviceToDevice);
        history[i] = (float4 *)malloc(bytes4);
        hipMemcpy(history[i], nPos, bytes4, hipMemcpyDeviceToHost);
    }

    // 计时
    clock_t end = clock();
    float elapsedTime = 1.0 * (end - start) / CLOCKS_PER_SEC;
    puts("Calculation Completed.");
    printf("Elapsed Time = %fs\n", elapsedTime);
    puts("Result Output Start.");

    // 重定向输出
    freopen("data.out", "w", stdout);
    printf("%d %d\n", n, T);
    for (int i = 0; i < T; ++i)
    {
        for (int j = 0; j < n; ++j)
            printf("%f %f %f\n", history[i][j].x, history[i][j].y, history[i][j].z);
        puts("");
    }
    fclose(stdout);

    // 释放设备内存
    hipFree(dPos);
    hipFree(dVel);
    hipFree(nPos);

    // 释放主机内存
    free(hPos);
    free(hVel);
    for (int i = 0; i < T; ++i)
        free(history[i]);

    puts("Process End.");

    return 0;
}